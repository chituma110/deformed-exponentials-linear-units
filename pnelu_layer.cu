#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/pnelu_layer.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Dtype>
__global__ void PNELUForward(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* slope_data,
    const int div_factor, Dtype t) {

  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out[index] = in[index] > 0 ? in[index] : ( pow( (1+(1-t)*in[index]), (1/(1-t)) ) - 1) * slope_data[c];
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void PNELUBackward(const int n, const int channels, const int dim,
    const Dtype* out_data, const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* slope_data, const int div_factor, Dtype t) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * (pow((out_data[index] + slope_data[c]),t) * pow(slope_data[c], (1-t))) );
  }
}

// CUDA kernel for element-wise parameter backward
template <typename Dtype>
__global__ void PNELUParamBackward(const int n,
    const int rows, const int rowPitch, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype t) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ( pow( (1+(1-t)*in_data[index]), (1/(1-t)) ) - 1) * (in_data[index] <= 0);
    for ( int k = 1; k < rows; k++ ) {
        out_diff[index] += in_diff[index + k*rowPitch]
           * ( pow( (1+(1-t)*in_data[index + k*rowPitch]), (1/(1-t)) ) - 1) * (in_data[index + k*rowPitch] <= 0);
        
    }
  }
}

template <typename Dtype>
void PNELULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  const Dtype* slope_data = this->blobs_[0]->gpu_data();
  const int div_factor = channel_shared_ ? channels : 1;
  Dtype t = this->layer_param_.pnelu_param().t();
  if (top[0] == bottom[0]) {
    caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  PNELUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, channels, dim, bottom_data, top_data, slope_data, div_factor, t);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void PNELULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  Dtype t = this->layer_param_.pnelu_param().t();

  // For in-place computation
  if (top[0] == bottom[0]) {
    bottom_data = bottom_memory_.gpu_data();
  }

  // Propagate to param
  // Since to write bottom diff will affect top diff if top and bottom blobs
  // are identical (in-place computaion), we first compute param backward to
  // keep top_diff unchanged.
  if (this->param_propagate_down_[0]) {
    Dtype* slope_diff = this->blobs_[0]->mutable_gpu_diff();
    int cdim = channels * dim;

    // compute element-wise diff
    // NOLINT_NEXT_LINE(whitespace/operators)
    PNELUParamBackward<Dtype><<<CAFFE_GET_BLOCKS(cdim),
      CAFFE_CUDA_NUM_THREADS>>>(
      cdim, bottom[0]->num(), top[0]->offset(1), top_diff ,
      bottom_data ,
      backward_buff_.mutable_gpu_diff(), t);
    CUDA_POST_KERNEL_CHECK;
    if (channel_shared_) {
      Dtype dsum;
      caffe_gpu_dot<Dtype>(channels * dim, backward_buff_.gpu_diff(),
       multiplier_.gpu_data(), &dsum);
      caffe_gpu_add_scalar(this->blobs_[0]->count(), Dtype(dsum), slope_diff);
    } else {
      caffe_gpu_gemv<Dtype>(CblasNoTrans, channels, dim, 1.,
        backward_buff_.gpu_diff(), multiplier_.gpu_data(), 1.,
        slope_diff);
    }
  }
  // Propagate to bottom
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* slope_data = this->blobs_[0]->gpu_data();
    int div_factor = channel_shared_ ? channels : 1;
    // NOLINT_NEXT_LINE(whitespace/operators)
    PNELUBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
        count, channels, dim, top_data, top_diff, bottom_data, bottom_diff, slope_data,
        div_factor, t);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(PNELULayer);


}  // namespace caffe
